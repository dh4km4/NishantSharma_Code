/*
Author: Nishant Sharma
Class: ECE4122
Last Date Modified: 11/20/2021
Description: 
The purpose of this file is to solve the 2D Steady
State Heat conduction in a thin plate problem, using 
iteration. It uses the Cuda library, and uses the gpu 
to do the computation.   

Running Procedure and Clarifications:- 
Getting a Pace ice gpu job with one gpu, 
setting up VNC, and running:
module load gcc/9.2.0 cuda/11.1
nvcc *.cu

Only the milliseconds is printed in the console without the units
as per the instructions and example on piazza.

References and links:-
https://docs.nvidia.com/cuda/cuda-c-programming-guide/index.html (Cuda Examples and guide)
https://stackoverflow.com/questions/7876624/timing-cuda-operations (Using cudaEvent for timing)
https://cpp.hotexamples.com/examples/-/-/cudaGetDeviceProperties/cpp-cudagetdeviceproperties-function-examples.html (cuda getDeviceProperties)

*/

#include <stdio.h>
#include <iostream>
#include <iomanip>
#include <fstream>
#include <hip/hip_runtime.h>

using namespace std;

/**
 * The Heat Compute function get the average of surrounding 
 * elements in the sheet and setting them in the copy array. 
 * This array contains the new values of the interior points in the sheet
 * and they will be copied over to the original array in the next method (Jacobian)
 *
 * Input Arguments: Sheet and Copy arrays, numElements = (N+1)^2 (Total number of elements), and N (Interior row width)
 */
__global__ void heatCompute(double *sheet, double *copy, int numElements, int N) 
{
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    int rowWidth = numElements / (N + 2);
    if (tid <= (numElements)) {
        int i = tid % rowWidth;
        int j = tid / rowWidth;
        if (i > 0 && i <= N && j > 0 && j <= N)
        {
            copy[(N+2) * i + j] = 0.25 * (sheet[(N+2) * (i - 1) + (j)] + sheet[(N + 2) * (i + 1) + (j)] + sheet[(N + 2) * (i) + (j - 1)] + sheet[(N + 2) * (i) + (j + 1)]);
        }
    }
}

/**
 * Jacobian Function sets the original sheet variable, which 
 * contains the main 2D array to the values in copy, which are
 * the updated averages. 
 * 
 * Input Arguments: Sheet and Copy array, numElemetns: (N+2) * (N+2), N: Has the interior row
 */
__global__ void jacobian(double *sheet, double *copy, int numElements, int N) 
{
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    int rowWidth = numElements / (N + 2);
    if (tid <= (numElements)) {
        int i = tid % rowWidth;
        int j = tid / rowWidth;
        if (i > 0 && i <= N && j > 0 && j <= N)
        {
            sheet[(N + 2) * i + j] = copy[(N + 2) * i + j];
        }
    }
}

/**
 * Host main function, takes in the input arguments
 * for N and I, where N is the interior row width of the sheet, and 
 * I is the number of iterations to run the algorithm. Error checking 
 * is done, such that only positive values for -N and -I as passed. 
 * Finally the arrays are created copied over to the GPU, and the GPU 
 * routines are called. 
 */
int main(int argc, char * argv[])
{
    int N = 0;
    int I = 0;
    if (argc != 5)
    {
      cout << "Invalid parameters, please check your values." << endl;
      return 0;
    }
    if (strcmp("-N", argv[1]) != 0)
    {
      cout << "Invalid parameters, please check your values." << argv[1] << endl;
      return 0;
    }
    if (strcmp("-I", argv[3]) != 0)
    {
      cout << "Invalid parameters, please check your values." << endl;
      return 0;
    }
    try
    {
      N = atoi(argv[2]);
      I = atoi(argv[4]);
    }
    catch (exception & e)
    {
      cout << "Invalid parameters, please check your values." << endl;
      return 0;
    }
    if (N <= 0 || I <= 0)
    {
      cout << "Invalid parameters, please check your values." << endl;
      return 0;
    }

    // Creating the host sheet and copy array
    size_t size = (N + 2) * (N + 2) * sizeof(double);
    double *h_sheet = (double *)malloc(size);
    double *h_copy = (double *)malloc(size);
    int numElements = (N+2)*(N+2);

    // Initialization
    for (int i = 0; i < numElements; i++)
    {
        if ((i > 0.3 * (N + 2 - 1)) && (i < 0.7 * (N + 2 - 1)))
        {
            h_sheet[i] = 100.0;
        }
        else
        {
            h_sheet[i] = 20.0;
        }
        h_copy[i] = 0.0;
    }
    
    // Device sheet and copy array pointers created and data has been copied
    double *d_sheet = NULL;
    hipMalloc(&d_sheet, size);
    
    double *d_copy = NULL;
    hipMalloc(&d_copy, size);
    
    hipMemcpy(d_sheet, h_sheet, size, hipMemcpyHostToDevice);
    hipMemcpy(d_copy, h_copy, size, hipMemcpyHostToDevice);

    // Getting maximum number of threads and running setting 
    // thread and block number
    int dev = 0;
    hipGetDevice(&dev);
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, dev);
    if (prop.maxThreadsPerBlock <= 0)
    {
        cout << "Error in Running, invalid max number of threads" << endl;
    }
    int threadsPerBlock = prop.maxThreadsPerBlock;
    int blocksPerGrid = (numElements + threadsPerBlock - 1) / threadsPerBlock;    
    

    // Recording time
    float time;
    hipEvent_t start, stop;

    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    for(int iteration = 0; iteration < I; iteration++)
    {
        // Does it synchronously - Updates all first, then set it equal
        heatCompute<<<blocksPerGrid, threadsPerBlock>>>(d_sheet, d_copy, numElements, N);
        jacobian<<<blocksPerGrid, threadsPerBlock>>>(d_sheet, d_copy, numElements, N);
    }

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);
    cout << fixed << setprecision(2) << time << endl;
    
    // Copy answer back from device to host
    hipMemcpy(h_sheet, d_sheet, size, hipMemcpyDeviceToHost);

    // Printing to csv
    ofstream outputFile;
    outputFile.open("finalTemperatures.csv", ios::out);
    outputFile << fixed << setprecision(6);
    for (int i = 0; i <= N+1; i++)
    {
        for (int j = 0; j <= N+1; j++)
        {
            outputFile << h_sheet[(N + 2) * i + j] << ",";
        }
        outputFile << "\n";
    }
    outputFile.close();

    hipFree(d_sheet);
    hipFree(d_copy);

    // Free host memory
    free(h_sheet);
    free(h_copy);
    return 0;
}
